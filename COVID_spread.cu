//
//  COVID_spread.c 
//  Parallel Programming, Spring 2020 
//

#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>



//use random number generator to initialize the Graph
//this init function is after the initialization of MPI
//initialize Graph, G_data, G_resultdata and Day_of_cure

// Global function #1: Initialization 
__global__ void gol_init(const int population, unsigned int** Graph, unsigned int* G_data, unsigned int* G_resultdata, unsigned int * Day_of_cure, unsigned int* Num_of_connections_per_person, int max_connections ){
		int i;
		// Initialize G_data, G_resultData and Day_of_cure on the CPU/GPU
		for (i = 0; i < population; i ++){
			G_data[i] = 0;
			G_resultdata[i] = 0;
			Day_of_cure[i] = -1; 
		}
	  
		return;
}


extern "C" void gol_init_master(const int population, unsigned int** Graph, unsigned int* G_data, unsigned int* G_resultdata, unsigned int * Day_of_cure, int myrank, unsigned int* Num_of_connections_per_person, int threadsCount, int max_connections ){

    int N=population;
    int numBlocks = (N+threadsCount-1) / threadsCount;
		
		// Initialize memory 
		hipMallocManaged(&Graph, (population * sizeof(unsigned int*)));
		hipMallocManaged(&G_data, (population * sizeof(unsigned int)));
		hipMallocManaged(&G_resultdata, (population * sizeof(unsigned int)));
		hipMallocManaged(&Day_of_cure, (population * sizeof(unsigned int)));
		hipMallocManaged(&Num_of_connections_per_person, (population * sizeof(unsigned int)));
		
		int i;
		for (i = 0; i < population; i++){
			int num_connections = rand() % max_connections + 1; 
			Num_of_connections_per_person[i] = num_connections;
			hipMallocManaged(&Graph[i], (num_connections * sizeof(unsigned int)));
		}
		
		// Initialize Graph on the CPU/GPU
		for (i = 0; i < population; i++){
				// Generate connections for this indiviudal
				int j;
				for (j = 0; j < Num_of_connections_per_person[j]; j++){
						int curr_connection = rand() % population + 1;
						Graph[i][j] = curr_connection; 
				}			
		}	
		
    gol_init<<<numBlocks, threadsCount>>>(population,Graph,G_data,G_resultdata,Day_of_cure,Num_of_connections_per_person, max_connections);
    
    int cE, cudaDeviceCount;
    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
      printf(" Unable to determine cuda device count, error is %d, count is %d\n",
             cE, cudaDeviceCount );
      exit(-1); 
    }
	
    if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess )
    {
      printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
             myrank, (myrank % cudaDeviceCount), cE);
      exit(-1); 
    }
} 

//Generate G_resultdata from current G_data and current Day_of_cure, update Day_of_cure

// Global Function #2: Iteration 
__global__ void gol_kernel(const unsigned int* G_data, unsigned int* G_resultData, unsigned int** Graph, unsigned int* Day_of_cure, int threshold, int threadsCount, int currDay, const int population, unsigned int* Num_of_connections_per_person, const int recovery_period, unsigned int* invunerable_individuals){
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	
	// 1 -> invunerable
	// 2 -> vunerable 
	
	// Cure individuals that recover on this day 
	int j;
	for (j = index; j < population; j += blockDim.x * gridDim.x){
		
		// If the individual is infected 
		if (G_data[j] == 1){
			// In the case today is the day this individual is to be cured 
			if (Day_of_cure[j] == currDay){
		
				G_resultData[j] = 0;
				invunerable_individuals[j] = 1;
			}		
		}
	}
	
	// Process individuals that are infected 
	for (j = index; j < population; j+= blockDim.x * gridDim.x){
		
		// In the case that the current indiviudal is infected 
		if (G_data[j] == 1){
			
			// Process his/her connections and note its spread 
			int i;
			for (i = 0; i < Num_of_connections_per_person[j]; i++){
				
				// If the connection is not currently infected 
				if (G_data[i] == 0 && invunerable_individuals[i] != 1){
					
					G_resultData[i] = 1;
					Day_of_cure[i] = currDay + recovery_period;
				}				
			}
		}
	}
	
	return;
}

//count the number of infected people in one iteration
__global__ void countInfectedPeople (unsigned int* G_data, const int population, int* count){

	int i;
	int index = blockIdx.x *blockDim.x + threadIdx.x;
	for (i=index;i<population;i+=blockDim.x * gridDim.x){
		if (G_data[i]==1) *count++;
	}
	return; 
}

extern "C"  int getInfectedPeople(unsigned int* G_data, const int population, int threadsCount){
	int N=population;
	int numBlocks= (N+threadsCount-1)/threadsCount;
	int count = 0;
	countInfectedPeople<<<numBlocks, threadsCount>>>(G_data, population, &count);
	return count;
}

extern "C"  void gol_swap( unsigned int **G_data, unsigned int **G_resultdata)
{
    // You write this function - it should swap the pointers of pA and pB.
    
    unsigned int * tmp;
    //reset all elements in pB to 0
    tmp=*G_data;
    *G_data=*G_resultdata;
    *G_resultdata=tmp;
}

//gol_kernelLaunch returns the number of iterations to let the number infected people be larger than threshold
extern "C" int gol_kernelLaunch(unsigned int** G_data,
                              	unsigned int** G_resultData,
																unsigned int*** Graph,
                              	unsigned int** Day_of_cure,
			      										const int population,
                              	const int threshold,
                              	int threadsCount,
																unsigned int** Num_of_connections_per_person,
																const int recovery_period
														){

unsigned int* D_data = *G_data;
unsigned int* D_resultData = *G_resultData;
int N=population;
int numBlocks= (N+threadsCount-1)/threadsCount;

int infected = getInfectedPeople(D_data,population,threadsCount);

int i=0;
    
    while(infected<=threshold)
    {
				unsigned int* invunerable_individuals;
				hipMallocManaged(&invunerable_individuals, population * sizeof(unsigned int));
				int j;
				for (j = 0; j < population; j++){
					invunerable_individuals[j] = 0;
				}
				
			
        gol_kernel<<<numBlocks, threadsCount>>>(D_data, D_resultData, *Graph, *Day_of_cure, threshold, threadsCount, i, population, *Num_of_connections_per_person, recovery_period, invunerable_individuals);
        infected=getInfectedPeople(D_resultData,population,threadsCount);
        hipDeviceSynchronize();
				gol_swap(&D_data, &D_resultData);
				i++;
  
				hipFree(&invunerable_individuals);
				
		}
		
		G_data = &D_data;
		G_resultData = &D_resultData;
		
    hipDeviceSynchronize();
    return i;
}



// print the indexes of infected people when the number of infected people is larger than threshold

extern "C"  void gol_print_infected(unsigned int* G_data, const int population)
{
    int i;
    
    for( i = 0; i < population; i++)
    {
        
        printf("%d ",G_data[i]);
	//print 100 indexes in each line
	if (i+1% 100==0){
	    printf("\n");
	} 
    }
    
    printf("\n\n");
}

extern "C"  void gol_print_cured(unsigned int** Day_of_cure, const int population){


}


