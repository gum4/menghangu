#include "hip/hip_runtime.h"
//
//  COVID_spread.cu
//  Parallel Programming, Spring 2020 
//

#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>



//use random number generator to initialize the Graph
//this init function is after the initialization of MPI
//initialize Graph, G_data, G_resultdata and Day_of_cure

// Global function #1: Initialization 
__global__ void gol_init(const int population, const int max_connections, unsigned int** Graph, unsigned int* G_data, unsigned int* G_resultdata, unsigned int * Day_of_cure ){

		// Initialize Graph on the CPU/GPU
		hipMallocManaged(Graph, (population * sizeof(unsigned int*)));
		int i;
		for (i = 0; i < population; i++){
			
				// Generate # of connections for this individual 
				int num_connections = rand() % max_connections + 1; 
				 
				// Generate connections for this indiviudal
				hipMallocManaged(Graph[i], (num_connections * sizeof(unsigned int)));
				int j;
				for (j = 0; j < num_connections; j++){
						int curr_connection = rand() % population + 1;
						Graph[i][j] = curr_connection; 
				}			
		}
		
		// Initialize G_data, G_resultData and Day_of_cure on the CPU/GPU
		hipMallocManaged(G_data, (population * sizeof(unsigned int)));
		hipMallocManaged(G_resultData, (population * sizeof(unsigned int)));
		hipMallocManaged(Day_of_cure, (population * sizeof(unsigned int)));
		for (i = 0; i < population; i ++){
			G_data[i] = 0;
			G_resultData = 0;
			Day_of_cure = -1; 
		}
	  
		return;
}


extern "C" void gol_init_master(const int population, unsigned int** Graph, unsigned int* G_data, unsigned int* G_resultdata, unsigned int ** Day_of_cure, int myrank ){

	
    int N=population;
    int numBlocks = (N+threadsCount-1) / threadsCount;
    gol_init<<<numBlocks, threadsCount>>>(population,Graph,G_data,G_resultdata,Day_of_cure);
    
    
    int cE, cudaDeviceCount;
    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
      printf(" Unable to determine cuda device count, error is %d, count is %d\n",
             cE, cudaDeviceCount );
      exit(-1); 
    }
	
    if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess )
    {
      printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
             myrank, (myrank % cudaDeviceCount), cE);
      exit(-1); 
    }
} 

//Generate G_resultdata from current G_data and current Day_of_cure, update Day_of_cure

__global__ void gol_kernel(const unsigned int* G_data,
                                   unsigned int** Day_of_cure,
                                   unsigned int* G_resultData){

	

}

//count the number of infected people in one iteration
__global__ int  countInfectedPeople (unsigned int* G_data, int population){

	int i;
	int count=0;
	int index = blockIdx.x *blockDim.x + threadIdx.x;
	for (i=index;i<population;i+=blockDim.x * gridDim.x){
		if (G_data[i]==1) count++;
	}
	return count;
}


extern "C"  int getInfectedPeople(unsigned int* G_data, int population, ushort threads count){
	int N=population;
	int numBlocks= (N+threadsCount-1)/threadsCount;
	int infected = countInfectedPeople<<<numBlocks, threadsCount>>>(G_data, population);
	return infected;
}

extern "C"  void gol_swap( unsigned int **G_data, unsigned int **G_resultdata)
{
    // You write this function - it should swap the pointers of pA and pB.
    
    unsigned int * tmp;
    //reset all elements in pB to 0
    tmp=*G_data;
    *G_data=*G_resultdata;
    *G_resultdata=tmp;
}

//gol_kernelLaunch returns the number of iterations to let the number infected people be larger than threshold
extern "C" int gol_kernelLaunch(unsigned int** G_data,
                              unsigned int** G_resultData,
                              unsigned int** Day_of_cure,
			      const int population,
                              const int threshold,
                              ushort threadsCount){

unsigned int* D_data = & G_data;
unsigned int* D_resultData = & G_resultData;
int N=population;
int numBlocks= (N+threadsCount-1)/threadsCount;

int infected = getInfectedPeople(D_data,population,threadscount);

size_t i=0;
    
    while(infected<=threshold)
    {
                
	// swap
        
        gol_kernel<<<numBlocks, threadsCount>>>(D_data, D_resultData, Day_of_cure, threshold, threadsCount);
        infected=getInfectedPeople(D_resultdata,population,threadscount);
        hipDeviceSynchronize();
	gol_swap(G_data, G_resultData);
	i++;
    }
    hipDeviceSynchronize();
    return i;
}



// print the indexes of infected people when the number of infected people is larger than threshold

extern "C"  void gol_print_infected(unsigned int* G_data, int population)
{
    int i;
    
    for( i = 0; i < population; i++)
    {
        
        printf("%d ",G_data[i]);
	//print 100 indexes in each line
	if (i+1% 100==0){
	    printf("\n");
	} 
    }
    
    printf("\n\n");
}

extern "C"  void gol_print_cured(unsigned int** Day_of_cure, int population){


}


