#include "hip/hip_runtime.h"
//
//  COVID_spread.c 
//  Parallel Programming, Spring 2020 
//

#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>



//use random number generator to initialize the Graph
//this init function is after the initialization of MPI
//initialize Graph, G_data, G_resultdata and Day_of_cure

// Global function #1: Initialization 
__global__ void gol_init(const int population, const int max_connections, unsigned int** Graph, unsigned int* G_data, unsigned int* G_resultdata, unsigned int * Day_of_cure, unsigned int* Num_of_connections_per_person ){

		// Initialize Graph on the CPU/GPU
		hipMallocManaged(Graph, (population * sizeof(unsigned int*)));
		int i;
		for (i = 0; i < population; i++){
			
				// Generate # of connections for this individual 
				int num_connections = rand() % max_connections + 1; 
				
				// Store the # of connections this individual has 
				Num_of_connections_per_person[i] = num_connections;
				
				// Generate connections for this indiviudal
				hipMallocManaged(Graph[i], (num_connections * sizeof(unsigned int)));
				int j;
				for (j = 0; j < num_connections; j++){
						int curr_connection = rand() % population + 1;
						Graph[i][j] = curr_connection; 
				}			
		}
		
		// Initialize G_data, G_resultData and Day_of_cure on the CPU/GPU
		hipMallocManaged(G_data, (population * sizeof(unsigned int)));
		hipMallocManaged(G_resultData, (population * sizeof(unsigned int)));
		hipMallocManaged(Day_of_cure, (population * sizeof(unsigned int)));
		for (i = 0; i < population; i ++){
			G_data[i] = 0;
			G_resultData = 0;
			Day_of_cure = -1; 
		}
	  
		return;
}


extern "C" void gol_init_master(const int population, unsigned int** Graph, unsigned int* G_data, unsigned int* G_resultdata, unsigned int * Day_of_cure, int myrank, unsigned int* Num_of_connections_per_person ){

	
    int N=population;
    int numBlocks = (N+threadsCount-1) / threadsCount;
    gol_init<<<numBlocks, threadsCount>>>(population,Graph,G_data,G_resultdata,Day_of_cure,Num_of_connections_per_person);
    
    
    int cE, cudaDeviceCount;
    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
      printf(" Unable to determine cuda device count, error is %d, count is %d\n",
             cE, cudaDeviceCount );
      exit(-1); 
    }
	
    if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess )
    {
      printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
             myrank, (myrank % cudaDeviceCount), cE);
      exit(-1); 
    }
} 

//Generate G_resultdata from current G_data and current Day_of_cure, update Day_of_cure

// Global Function #2: Iteration 
__global__ void gol_kernel(const unsigned int* G_data, unsigned int* G_resultData, unsigned int** Graph, unsigned int* Day_of_cure, int threshold, int threadsCount, int currDay, int population){
	
	int index = blockId.x * blockDim.x + threadIdx.x;
	
	unsigned int invunerable_individuals[population] = {0};
	// 1 -> invunerable
	// 2 -> vunerable 
	
	// Cure individuals that recover on this day 
	int j;
	for (j = index; j < population; j += blockDim.x * gridDim.x){
		
		// If the individual is infected 
		if (G_data[j] == 1){
			// In the case today is the day this individual is to be cured 
			if (Day_of_cure[j] == currDay){
				G_resultData[j] = 0;
				invunerable_individuals[j] = 1;

			}		
		}
	}
	
	// Process individuals that are infected 
	for (j = index; j < population; j+= blockDim.x * gridDim.x){
		
		// In the case that the current indiviudal is infected 
		if (G_data[j] == 1){
			
			// Process his/her connections and note its spread 
			int i;
			for (i = 0; i < Num_of_connections_per_person[j]; i++){
				
				// If the connection is not currently infected 
				if (G_data[i] == 0 && invunerable_individuals[i] != 1){
					G_resultData[i] = 1;
					Day_of_cure[i] = currDay + // need variable for number of days to be cured here 
				}				
			}
		}
	}
	
	return;
}

//count the number of infected people in one iteration
__global__ int  countInfectedPeople (unsigned int* G_data, int population){

	int i;
	int count=0;
	int index = blockIdx.x *blockDim.x + threadIdx.x;
	for (i=index;i<population;i+=blockDim.x * gridDim.x){
		if (G_data[i]==1) count++;
	}
	return count;
}


extern "C"  int getInfectedPeople(unsigned int* G_data, int population, ushort threads count){
	int N=population;
	int numBlocks= (N+threadsCount-1)/threadsCount;
	int infected = countInfectedPeople<<<numBlocks, threadsCount>>>(G_data, population);
	return infected;
}

extern "C"  void gol_swap( unsigned int **G_data, unsigned int **G_resultdata)
{
    // You write this function - it should swap the pointers of pA and pB.
    
    unsigned int * tmp;
    //reset all elements in pB to 0
    tmp=*G_data;
    *G_data=*G_resultdata;
    *G_resultdata=tmp;
}

//gol_kernelLaunch returns the number of iterations to let the number infected people be larger than threshold
extern "C" int gol_kernelLaunch(unsigned int* G_data,
                              unsigned int* G_resultData,
															unsigned int** Graph,
                              unsigned int* Day_of_cure,
			      									const int population,
                              const int threshold,
                              ushort threadsCount,
															Num_of_connections_per_person){

unsigned int* D_data = & G_data;
unsigned int* D_resultData = & G_resultData;
int N=population;
int numBlocks= (N+threadsCount-1)/threadsCount;

int infected = getInfectedPeople(D_data,population,threadscount);

size_t i=0;
    
    while(infected<=threshold)
    {
                
	// swap
        
        gol_kernel<<<numBlocks, threadsCount>>>(D_data, D_resultData, Graph, Day_of_cure, threshold, threadsCount, i, population, Num_of_connections_per_person);
        infected=getInfectedPeople(D_resultdata,population,threadscount);
        hipDeviceSynchronize();
	gol_swap(G_data, G_resultData);
	i++;
    }
    hipDeviceSynchronize();
    return i;
}



// print the indexes of infected people when the number of infected people is larger than threshold

extern "C"  void gol_print_infected(unsigned int* G_data, int population)
{
    int i;
    
    for( i = 0; i < population; i++)
    {
        
        printf("%d ",G_data[i]);
	//print 100 indexes in each line
	if (i+1% 100==0){
	    printf("\n");
	} 
    }
    
    printf("\n\n");
}

extern "C"  void gol_print_cured(unsigned int** Day_of_cure, int population){


}


